#include "hip/hip_runtime.h"
/*
Attempt SummarySubmit Attempt for Grading
Remember to answer the questions before clicking.
Dataset Id:	5
Created:	less than a minute ago
Status:	Correct solution for this dataset.
Timer Output
Kind	Location	Time (ms)	Message
GPU	main.cu::109	8.189686	Doing GPU Computation (memory + compute)
GPU	main.cu::111	1.620635	Doing GPU memory allocation
Copy	main.cu::117	2.013498	Copying data to the GPU
Compute	main.cu::122	0.826587	Doing the computation on the GPU
Copy	main.cu::130	3.67117	Copying data from the GPU
*/

#include    <wb.h>

#define wbCheck(stmt)                                       \
    do {                                                    \
        hipError_t err = stmt;                             \
        if (err != hipSuccess) {                           \
            wbLog(ERROR, "Failed to run stmt ", #stmt);     \
            return -1;                                      \
        }                                                   \
    } while(0)

#define Mask_width  5
#define Mask_radius Mask_width / 2
#define TILE_WIDTH  16
#define SIZE        (TILE_WIDTH + Mask_width - 1)

//@@ INSERT CODE HERE
__global__
void Image_Convolution (float * I, const float * __restrict__ M, float * P, int channels, int width, int height)
{
    __shared__ float N_ds[SIZE][SIZE];

    int block_x = blockIdx.x,  block_y = blockIdx.y;
    int thread_x = threadIdx.x, thread_y = threadIdx.y;

    //lookup variable designations and formula uses + borrow loops from git
    for (int k = 0; k < channels; ++k) {
        int dest  = thread_y * TILE_WIDTH + thread_x;
        int destX = dest % SIZE;
        int destY = dest / SIZE;
        int srcY  = block_y * TILE_WIDTH + destY - Mask_radius;
        int srcX  = block_x * TILE_WIDTH + destX - Mask_radius;
        int src   = (srcY * width + srcX) * channels + k;

        if (srcY >= 0 && srcY < height && srcX >= 0 && srcX < width)
            N_ds[destY][destX] = I[src];
        else
            N_ds[destY][destX] = 0.0;

        dest  = thread_y * TILE_WIDTH + thread_x + TILE_WIDTH * TILE_WIDTH;
        destY = dest / SIZE;
        destX = dest % SIZE;
        srcY  = block_y * TILE_WIDTH + destY - Mask_radius;
        srcX  = block_x * TILE_WIDTH + destX - Mask_radius;
        src   = (srcY * width + srcX) * channels + k;

        if (destY < SIZE) {
            if (srcY >= 0 && srcY < height && srcX >= 0 && srcX < width)
                N_ds[destY][destX] = I[src];
            else
                N_ds[destY][destX] = 0.0;
        }
        __syncthreads();

        float accum = 0;
        for (int y = 0; y < Mask_width; ++y)
            for (int x = 0; x < Mask_width; ++x)
                accum += N_ds[thread_y + y][thread_x + x] * M[y * Mask_width + x];

        int x = block_x * TILE_WIDTH + thread_x;
        int y = block_y * TILE_WIDTH + thread_y;
        
        if (y < height && x < width)
            P[(y * width + x) * channels + k] = min(max(accum, 0.0), 1.0);

        __syncthreads();
    }
}

int main (int argc, char * argv[ ])
{
    wbArg_t arg;
    int maskRows;
    int maskColumns;
    int imageChannels;
    int imageWidth;
    int imageHeight;
    char * inputImageFile;
    char * inputMaskFile;
    wbImage_t inputImage;
    wbImage_t outputImage;
    float * hostInputImageData;
    float * hostOutputImageData;
    float * hostMaskData;
    float * deviceInputImageData;
    float * deviceOutputImageData;
    float * deviceMaskData;

    arg = wbArg_read(argc, argv); /* parse the input arguments */

    inputImageFile = wbArg_getInputFile(arg, 0);
    inputMaskFile = wbArg_getInputFile(arg, 1);

    inputImage = wbImport(inputImageFile);
    hostMaskData = (float *) wbImport(inputMaskFile, &maskRows, &maskColumns);

    assert(maskRows == 5); /* mask height is fixed to 5 in this mp */
    assert(maskColumns == 5); /* mask width is fixed to 5 in this mp */

    imageWidth = wbImage_getWidth(inputImage);
    imageHeight = wbImage_getHeight(inputImage);
    imageChannels = wbImage_getChannels(inputImage);

    outputImage = wbImage_new(imageWidth, imageHeight, imageChannels);

    hostInputImageData = wbImage_getData(inputImage);
    hostOutputImageData = wbImage_getData(outputImage);

    wbTime_start(GPU, "Doing GPU Computation (memory + compute)");

    wbTime_start(GPU, "Doing GPU memory allocation");
    hipMalloc((void **) &deviceInputImageData, imageWidth * imageHeight * imageChannels * sizeof(float));
    hipMalloc((void **) &deviceOutputImageData, imageWidth * imageHeight * imageChannels * sizeof(float));
    hipMalloc((void **) &deviceMaskData, maskRows * maskColumns * sizeof(float));
    wbTime_stop(GPU, "Doing GPU memory allocation");

    wbTime_start(Copy, "Copying data to the GPU");
    hipMemcpy(deviceInputImageData, hostInputImageData, imageWidth * imageHeight * imageChannels * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(deviceMaskData, hostMaskData, maskRows * maskColumns * sizeof(float), hipMemcpyHostToDevice);
    wbTime_stop(Copy, "Copying data to the GPU");

    wbTime_start(Compute, "Doing the computation on the GPU");
    //@@ INSERT CODE HERE
    dim3 dimGrid(ceil((float) imageWidth / TILE_WIDTH), ceil((float) imageHeight / TILE_WIDTH));
    dim3 dimBlock(TILE_WIDTH, TILE_WIDTH, 1);
    Image_Convolution<<<dimGrid, dimBlock>>>(deviceInputImageData, deviceMaskData, deviceOutputImageData, imageChannels, imageWidth, imageHeight);

    wbTime_stop(Compute, "Doing the computation on the GPU");

    wbTime_start(Copy, "Copying data from the GPU");
    hipMemcpy(hostOutputImageData, deviceOutputImageData, imageWidth * imageHeight * imageChannels * sizeof(float), hipMemcpyDeviceToHost);
    wbTime_stop(Copy, "Copying data from the GPU");

    wbTime_stop(GPU, "Doing GPU Computation (memory + compute)");

    wbSolution(arg, outputImage);

    hipFree(deviceInputImageData);
    hipFree(deviceOutputImageData);
    hipFree(deviceMaskData);

    free(hostMaskData);
    wbImage_delete(outputImage);
    wbImage_delete(inputImage);

    return 0;
}